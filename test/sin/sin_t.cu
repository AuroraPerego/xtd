#include "hip/hip_runtime.h"
#define CATCH_CONFIG_MAIN
#include <catch.hpp>

#include "math.h"
#include <hip/hip_runtime.h>
#include <limits>
#include <vector>

template <typename T> __global__ void sinKernel(double *result, T input) {
  result[0] = static_cast<double>(xtd::sin(input));
}

template <typename T> __global__ void sinfKernel(double *result, T input) {
  result[0] = static_cast<double>(xtd::sinf(input));
}

TEST_CASE("sinCuda", "[sin]") {
  int deviceCount;
  hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);

  if (cudaStatus != hipSuccess || deviceCount == 0) {
    exit(EXIT_SUCCESS);
  }

  hipSetDevice(0);
  hipStream_t q;
  hipStreamCreate(&q);

  // input
  std::vector<double> values{-1., 0., M_PI / 2, M_PI, 42.};

  double* result;
  int constexpr N = 6;
  hipMallocAsync(&result, N * sizeof(double), q);

  for (auto v : values) {

    hipMemsetAsync(&result, 0x00, N * sizeof(double), q);

    sinKernel<<<1, 1, 0, q>>>(&result[0], static_cast<int>(v));
    sinKernel<<<1, 1, 0, q>>>(&result[1], static_cast<float>(v));
    sinKernel<<<1, 1, 0, q>>>(&result[2], static_cast<double>(v));
    sinfKernel<<<1, 1, 0, q>>>(&result[3], static_cast<int>(v));
    sinfKernel<<<1, 1, 0, q>>>(&result[4], static_cast<float>(v));
    sinfKernel<<<1, 1, 0, q>>>(&result[5], static_cast<double>(v));

    double resultHost[N];
    hipMemcpyAsync(resultHost, result, N * sizeof(double), hipMemcpyDeviceToHost, q);

    hipStreamSynchronize(q);

    auto const epsilon = std::numeric_limits<double>::epsilon();
    auto const epsilon_f = std::numeric_limits<float>::epsilon();
    REQUIRE_THAT(resultHost[0], Catch::Matchers::WithinAbs(std::sin(static_cast<int>(v)), epsilon));
    REQUIRE_THAT(resultHost[1], Catch::Matchers::WithinAbs(std::sin(v), epsilon_f));
    REQUIRE_THAT(resultHost[2], Catch::Matchers::WithinAbs(std::sin(v), epsilon));
    REQUIRE_THAT(resultHost[3], Catch::Matchers::WithinAbs(sinf(static_cast<int>(v)), epsilon_f));
    REQUIRE_THAT(resultHost[4], Catch::Matchers::WithinAbs(sinf(v), epsilon_f));
    REQUIRE_THAT(resultHost[5], Catch::Matchers::WithinAbs(sinf(v), epsilon_f));
  }

  hipFreeAsync(result, q);
}
